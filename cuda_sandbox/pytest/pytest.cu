#include<stdio.h>
#include<hip/hip_runtime.h>



void errorGuard(hipError_t candidate){
	if (candidate != hipSuccess){
		fprintf(stderr, "%s\n", hipGetErrorString(candidate));
	}
}

__global__ void helloWorldGPU(){
	printf("IndexInGrid %d: Hello World\n", threadIdx.x + (blockIdx.x * blockDim.x));
}


// extern "C"{
// void hwg(){
int main(){
	helloWorldGPU<<<5,1>>>();
	hipDeviceSynchronize();
	exit(0);
// }
}
