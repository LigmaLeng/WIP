#include<stdio.h>
#include<stdlib.h>
#include<hipblas.h>

#define N (1<<10)
#define TILE_SIZE (1<<4)
#define SHMEM_SIZE (TILE_SIZE * TILE_SIZE * sizeof(int))

inline bool cudaGuard(hipError_t candidate){
    if (candidate != hipSuccess){
        fprintf("CUDA Runtime Error: %s\n", hipGetErrorString(candidate));
        return true;
    }
    return false;
}

inline bool cublasGuard(hipblasStatus_t candidate){
    if (candidate != HIPBLAS_STATUS_SUCCESS){
        fprintf("cuBLAS Runtime Error: %s\n", cublasGetStatusString(candidate));
        return true;
    }
    return false;
}


void initMatrices(int* a, int* b){
    for (int i = 1; i <= N; i++){
        for (int j = 1; j <= N; j++){
            // cuBLAS arrays arranged in Fortran Column Major order with 1-based indexing
        }
    }
}

int main(){
    //TODO
    int *a, *b, *c, *device_a, *device_b, *device_c;
    size_t size = (N) * (N) * sizeof(int);
    hipblasHandle_t handle;

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    hipMalloc(&device_a, size);
    hipMalloc(&device_b, size);
    hipMalloc(&device_c, size);

    initMatrices(a, b);
}
